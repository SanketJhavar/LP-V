#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
using namespace std;
void Intialize(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 10;
    }
}
void print(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        cout << a[i] << " ";
    }
    cout << endl;
}
__global__ void add(int *a, int *b, int *c, int n)
{
    int tt = blockIdx.x * blockDim.x + threadIdx.x;
    if (tt < n)
    {
        c[tt] = a[tt] + b[tt];
    }
}
int main()
{
    int n = 5;
    int vectorsize = n;
    size_t vectorByte = vectorsize * sizeof(n);

    int *a, *b, *c;
    a = new int[vectorsize];
    b = new int[vectorsize];
    c = new int[vectorsize];

    Intialize(a, n);
    Intialize(b, n);
    print(a, n);
    print(b, n);

    int *x, *y, *z;
    hipMalloc(&x, vectorByte);
    hipMalloc(&y, vectorByte);
    hipMalloc(&z, vectorByte);

    hipMemcpy(x, a, vectorByte, hipMemcpyHostToDevice);
    hipMemcpy(y, b, vectorByte, hipMemcpyHostToDevice);

    int thread = 256;
    int block = (n + thread - 1) / 256;
    add<<<block, thread>>>(x, y, z, n);
    hipMemcpy(c, z, vectorByte, hipMemcpyDeviceToHost);
    
    print(c,n);
    return 0;
}
