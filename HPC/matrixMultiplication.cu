#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
using namespace std;
__global__ void mulitply(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n and col < n)
    {
        int sum = 0;
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}
void Intialize(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 10;
    }
}
void print(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << a[i * n + j] << " ";
        }
        cout << endl;
    }
}
int main()
{
    int n = 2;
    int matrixsize = n * n;
    size_t metrixByte = matrixsize * sizeof(int);

    int *a, *b, *c;
    a = new int[matrixsize];
    b = new int[matrixsize];
    c = new int[matrixsize];

    Intialize(a, n);
    Intialize(b, n);

    print(a, n);
    print(b, n);
    int *x, *y, *z;

    hipMalloc(&x, metrixByte);
    hipMalloc(&y, metrixByte);
    hipMalloc(&z, metrixByte);

    hipMemcpy(x,a,metrixByte,hipMemcpyHostToDevice);
    hipMemcpy(y,b,metrixByte,hipMemcpyHostToDevice);

    int thread = 2;
    int block = n/thread;

    mulitply<<<block,thread>>>(x,y,z,n);
    hipMemcpy(c,z,metrixByte,hipMemcpyDeviceToHost);
    print(c,n);
    return 0;
}
